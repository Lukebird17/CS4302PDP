#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for matrix multiplication
__global__ void matrixMultiply(float *A, float *B, float *C, int N) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * blockDim.y + ty;
    int col = blockIdx.x * blockDim.x + tx;
    
    __shared__ float shared_A[32][32];  // Shared memory for sub-matrix A
    __shared__ float shared_B[32][32];  // Shared memory for sub-matrix B

    float C_value = 0.0f;
    
    // Loop over sub-matrices to compute the product
    for (int k = 0; k < (N / 32); ++k) {
        shared_A[ty][tx] = A[row * N + (k * 32 + tx)];
        shared_B[ty][tx] = B[(k * 32 + ty) * N + col];
        __syncthreads();

        for (int n = 0; n < 32; ++n) {
            C_value += shared_A[ty][n] * shared_B[n][tx];
        }
        __syncthreads();
    }
    
    if (row < N && col < N) {
        C[row * N + col] = C_value;
    }
}

void matrixMultiplyCUDA(float *A, float *B, float *C, int N) {
    // Allocate memory on the GPU
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Set up the grid and block sizes
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(N / 32, N / 32);

    // Launch the kernel
    matrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
